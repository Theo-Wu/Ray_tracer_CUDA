#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <float.h>
#include "common.h"
#include "sphere.h"
#include "cylinder.h"
#include "hitable_list.h"
#include "vec3.h"
#include "camera.h"
#include "material.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line){
    if(result){
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

#define RANDVEC3 vec3(hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state))


__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state){
    ray cur_ray = r;
    vec3 cur_throughput = vec3(1,1,1);
    for(int i = 0; i < 50; i++){
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            // return 0.5f*vec3(rec.normal.x()+1.0f, rec.normal.y()+1.0f, rec.normal.z()+1.0f); //normal
            ray scatter_ray;
            vec3 throughput;
            if(rec.material_pointer->scatter(cur_ray,rec,throughput,scatter_ray,local_rand_state)){
                cur_throughput *= throughput;
                cur_ray = scatter_ray;
            }else{
                return vec3(0,0,0);
            }
        }
        else { // sky
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_throughput * c;
        }
    }
    return vec3(0,0,0);
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)){
        return ;
    }
    int pixel_index = j*max_x + i;
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)){
        return ;
    }
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0,0,0);
    for(int s = 0; s < ns; s++){
        float u = float(i+hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j+hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u,v);
        col += color(r,world,&local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list)   = new sphere(vec3(0,0,-1), 0.5, new diffuse(vec3(0.8,0.3,0.3)));
        *(d_list+1) = new sphere(vec3(0,-100.5,-1), 100, new diffuse(vec3(0.8,0.8,0.0)));
        *(d_list+2) = new sphere(vec3(1,0,-1), 0.5, new metal(vec3(0.8,0.6,0.2),1.0));
        *(d_list+3) = new sphere(vec3(-1,0,-1),0.5, new dielectric(1.5));
        *(d_list+4) = new sphere(vec3(-1,0,-1),-0.45, new dielectric(1.5));
        *(d_list+5) = new sphere(vec3(0,0.5,-3.5), 2., new metal(vec3(0.2,0.6,0.8),0.0));
        // *(d_list+4) = new cylinder(vec3(-1.5,-0.2,-2),unit_vector(vec3(1.0,1.0,0)),0.3, new metal(vec3(0.8,0.8,0.8),0.3));
        *d_world    = new hitable_list(d_list,6);
        *d_camera = new camera();
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    delete *(d_list);
    delete *(d_list+1);
    delete *(d_list+2);
    delete *(d_list+3);
    delete *(d_list+4);
    delete *(d_list+5);
    delete *d_world;
    delete *d_camera;
}

int main(){
    // allocate an nx*ny image-sized frame buffer (FB) on the host 
    // to hold the RGB float values calculated by the GPU
    const int nx = 1200;
    const int ny = 600;
    const int ns = 100; //number of samples
    int tx = 8; //thread number
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << "blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));

    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 6*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks,threads>>>(nx,ny,d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks,threads>>>(fb,nx,ny,ns,d_camera,d_world,d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--){
        for (int i = 0; i < nx; i++){
            size_t pixel_index = j*nx + i;
            float r = fb[pixel_index].r();
            float g = fb[pixel_index].g();
            float b = fb[pixel_index].b();
            int ir = int(255.999*r);
            int ig = int(255.999*g);
            int ib = int(255.999*b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));
    hipDeviceReset();
}